#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel segmented scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
__global__ void upsweep_kernel(const int two_d, const int two_dplus1, int* input, int* output, const int N){
    unsigned int idx {blockIdx.x * blockDim.x + threadIdx.x};
    unsigned int k {two_dplus1 * idx};
    if (k < N){
        output[k + two_dplus1 - 1] = input[k + two_d - 1] + output[k + two_dplus1 - 1];
    }
}

__global__ void downsweep_kernel(const int two_d, const int two_dplus1, int* input, int* output, const int N){
    unsigned int idx {blockIdx.x * blockDim.x + threadIdx.x};
    unsigned int k {two_dplus1 * idx};
    if (k < N) {
        int tmp {input[k + two_d - 1]};
        output[k + two_d - 1] = input[k + two_dplus1 - 1];
        output[k + two_dplus1 - 1] = tmp + input[k + two_dplus1 - 1];
    }
}

void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep input
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
    
    // input and result are on device
    hipMemcpy(result, input, N * sizeof(int), hipMemcpyDeviceToDevice);

    // Up Sweep phase
    int rounded_length = nextPow2(N);
    for (int two_d = 1; two_d <= rounded_length / 2; two_d <<= 1) {
        int two_dplus1 = (two_d << 1);
        int threads_per_block {std::min(rounded_length / two_dplus1, THREADS_PER_BLOCK)};
        int num_blocks {(rounded_length / two_dplus1 + threads_per_block - 1) / threads_per_block};
        // dispatch to CUDA
        upsweep_kernel<<<num_blocks, threads_per_block>>>(two_d, two_dplus1, result, result, rounded_length);
        hipDeviceSynchronize();
    }
    //Down Sweep pahse
    hipMemset(&result[rounded_length - 1], 0, sizeof(int));
    for (int two_d = rounded_length / 2; two_d >= 1; two_d >>= 1) {
        int two_dplus1 = (two_d << 1);
        int threads_per_block {std::min(rounded_length / two_dplus1, THREADS_PER_BLOCK)};
        int num_blocks {(rounded_length / two_dplus1 + threads_per_block - 1) / threads_per_block};
        // dispatch to CUDA
        downsweep_kernel<<<num_blocks, threads_per_block>>>(two_d, two_dplus1, result, result, rounded_length);
        hipDeviceSynchronize();
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of segmented scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;

    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found

__global__ void repetition_kernel(int* input, int* indicator, const int N) {
    unsigned int idx {blockIdx.x * blockDim.x + threadIdx.x};
    if (idx < N - 1 && input[idx] == input[idx + 1]) {
        indicator[idx] = 1;
    } else {
        indicator[idx] = 0;
    }
}

__global__ void gather_kernel(int* exclusive_scan_results, int* output, const int N, int* total_count) {
    unsigned int idx {blockIdx.x * blockDim.x + threadIdx.x};
    if (idx < N - 1) {
        if (exclusive_scan_results[idx] != exclusive_scan_results[idx + 1]) {
            output[exclusive_scan_results[idx]] = idx;
        }
    } else if (idx == N - 1) {
        *total_count = exclusive_scan_results[N - 1];
    }
}

int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    const int rounded_length {nextPow2(length)};

    // construct indicator array, whether input[i] == input[i + 1]
    int* device_indicator;
    hipMalloc(&device_indicator, rounded_length * sizeof(int));
    int threads_per_block {std::min(rounded_length, THREADS_PER_BLOCK)};
    int num_blocks {(rounded_length + THREADS_PER_BLOCK - 1) / threads_per_block};
    repetition_kernel<<<num_blocks, threads_per_block>>>(device_input, device_indicator, length);

    // exclusive scan on indicator array
    // to get device_exclusive_scan_results
    int* device_exclusive_scan_results;
    hipMalloc(&device_exclusive_scan_results, rounded_length * sizeof(int));
    exclusive_scan(device_indicator, length, device_exclusive_scan_results);
    hipFree(device_indicator);

    // get repetition points in array    
    int* device_repetition_count;
    hipMalloc(&device_repetition_count, sizeof(int));
    gather_kernel<<<num_blocks, threads_per_block>>>(device_exclusive_scan_results, device_output, length, device_repetition_count);
    hipFree(device_exclusive_scan_results);

    // return results
    int output_length;
    hipMemcpy(&output_length, device_repetition_count, sizeof(int), hipMemcpyDeviceToHost);
    hipFree(device_repetition_count);
    
    return output_length; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
